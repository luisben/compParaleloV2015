
#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS 64

__global__ void vecSum(int *a, int *b, int size){
	
	__shared__ int c[THREADS];
	int tid = blockDim.x*blockIdx.x+threadIdx.x;
	int stid = threadIdx.x;
	if(tid < size){
		c[stid] = a[tid];
		c[stid] += b[tid];
	}
		__syncthreads();
	if(tid < size){
		a[tid] = c[stid];
	}
	
}

int main(int argc, char* argv[]){

	//initialization code
	int size,threads,blocks;
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size = 2048*2048;
	blocks = size/THREADS;
	threads = THREADS;
	while(blocks*threads < size)
		blocks++;
	int totalSize = size * sizeof(int);
	
	int *a,*b,*dev_a,*dev_b;

	hipMalloc((void**)&dev_a,totalSize);
	hipMalloc((void**)&dev_b,totalSize);

	a = (int*) malloc(totalSize);
	b = (int*) malloc(totalSize);
	//end mallocs

	int idx;

	for(idx=0;idx<size;idx++){
		a[idx] = idx;
		b[idx] = idx*2;
	}
	
	//copy to dev
	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,totalSize,hipMemcpyHostToDevice);

	int iteration = 0;
	float avg_time = 0.0;
	for(iteration=0;iteration<10;iteration++){
	//call kernel and measure times
	hipEventRecord(start,0);
	vecSum<<<blocks,threads>>>(dev_a,dev_b,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i blocks of %i threads : %f \n",blocks,threads,total_time);
	avg_time+=total_time;
	}
	avg_time/=10.0;
	printf("average time for %i size vector mult is %f ",size,avg_time);
	//copy back and prints
	hipMemcpy(a,dev_a,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n a[%i]=%i\n",idx,a[idx]);
 
	//free
	free(a);
	free(b);

	hipFree(dev_a);
	hipFree(dev_b);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	return 0;
}



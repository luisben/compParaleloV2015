#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel(int *d, int n){
	
	__shared__ int s[64];
	int tid = threadIdx.x;
	int tr + n - tid - 1;
	s[tid] = d[tid]
	_synchthreads();
	d[tid] = s[tr];
}

int main(int argc, char* argv[]){

	//initialization code
	int size,threads;
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size = 1000;
	int totalSize = size * sizeof(int);
	
	int *a,*r,*d,dev_d;

	hipMalloc((void**)&dev_d,totalSize);

	a = (int*) malloc(totalSize);
	r = (int*) malloc(totalSize);
	d = (int*) malloc(totalSize);
	//end mallocs

	//problem specific
	int idx;

	for(idx=0;idx<size;idx++){
		a[idx] = idx;
		r[idx] = size-idx-1;
		d[idx] = 0;
	}
	
	//copy to dev
	hipMemcpy(dev_d,a,totalSize,hipMemcpyHostToDevice);
	
	int iteration = 0;
	float avg_time = 0.0;
	for(iteration=0;iteration<10;iteration++){
	//call kernel and measure times
	hipEventRecord(start,0);
	vecMult<<<blocks,threads>>>(dev_a,dev_b,dev_c,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i blocks of %i threads : %f \n",blocks,threads,total_time);
	avg_time+=total_time;
	}
	avg_time/=10.0;
	printf("average time for %i size vector mult is %f ",size,avg_time);
	//copy back and prints
	hipMemcpy(d,dev_d,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n a[%i]=%i\n",idx,c[idx]);
 
	for(idx = 0;idx < n; idx++)
		if(d[i] != r[i])
			printf(”Verificar- Hay un error”); 
 
	//free
	free(a);
	free(r);
	free(d);

	hipFree(dev_d);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	return 0;
}



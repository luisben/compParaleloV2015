
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *d, int n){
	
	__shared__ int s[64];
	int tid = threadIdx.x;
	int tr = n - tid - 1;
	s[tid] = d[tid];
	__syncthreads();
	d[tid] = s[tr];
}

int main(int argc, char* argv[]){

	//initialization code
	int size;
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size = 64;
	int totalSize = size * sizeof(int);
	
	int *a,*r,*d,*dev_d;

	hipMalloc((void**)&dev_d,totalSize);

	a = (int*) malloc(totalSize);
	r = (int*) malloc(totalSize);
	d = (int*) malloc(totalSize);
	//end mallocs

	//problem specific
	int idx;

	for(idx=0;idx<size;idx++){
		a[idx] = idx;
		r[idx] = size-idx-1;
		d[idx] = 0;
	}
	
	//copy to dev
	hipMemcpy(dev_d,a,totalSize,hipMemcpyHostToDevice);
	
	int iteration = 0;
	float avg_time = 0.0;
	for(iteration=0;iteration<1;iteration++){
	//call kernel and measure times
	hipEventRecord(start,0);
	kernel<<<1,64>>>(dev_d,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i threads : %f \n",size,total_time);
	avg_time+=total_time;
	}
	avg_time/=10.0;
	printf("average time for %i size vector mult is %f ",size,avg_time);
	//copy back and prints
	hipMemcpy(d,dev_d,totalSize,hipMemcpyDeviceToHost);
 
	for(idx = 0;idx < size; idx++)
		if(d[idx] != r[idx])
			printf("Verificar- Hay un error"); 
 
	//free
	free(a);
	free(r);
	free(d);

	hipFree(dev_d);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	return 0;
}



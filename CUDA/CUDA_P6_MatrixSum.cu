
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecMatSum(int *a, int *b, int *c, int width, int length){
	
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int col = blockIdx.y*blockDim.y + threadIdx.y;
	int tid = row*width+col;
	if(tid < length)
		c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]){

	//initialization code
	int width,size,threads,blocks,totalSize;
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	if(argv[2])
		width = atoi(argv[2]);
	else 
		width = 300;
	size = width*width;
	if(argv[1])
		threads = atoi(argv[1]);
	else
		threads = 16;
	dim3 ThreadsInBlock(threads,threads); //will provide threads * threads threads
	blocks = (int) sqrt((float) size / (float) (threads*threads));
	dim3 BlockDim(blocks,blocks);
	while(BlockDim.x*BlockDim.y*threads*threads < size)
		BlockDim.y += 1;
	totalSize = size*sizeof(int);
	printf("\n%ix%i blocks of %ix%i threads = %i threads total \n",BlockDim.x,BlockDim.y,ThreadsInBlock.x,ThreadsInBlock.y,BlockDim.x*BlockDim.y*ThreadsInBlock.x*ThreadsInBlock.y);
	//end init
	
	//start mallocs
	int *a,*dev_a,*b,*dev_b,*c,*dev_c;

	hipMalloc((void**)&dev_a,totalSize);
	hipMalloc((void**)&dev_b,totalSize);
	hipMalloc((void**)&dev_c,totalSize);

	a = (int*) malloc(totalSize);
	b = (int*) malloc(totalSize);
	c = (int*) malloc(totalSize);
	//end mallocs

	//problem specific
	int idx;

	for(idx=0;idx<size;idx++){
		a[idx] = idx;
		b[idx] = idx*2;
	}
	
	//copy to dev
	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,totalSize,hipMemcpyHostToDevice);
	//end copy
	
	int iteration = 0;
	float avg_time = 0;
	for(iteration=0;iteration<10;iteration++){
	//call kernel and measure times
	hipEventRecord(start,0);
	vecMatSum<<<BlockDim,ThreadsInBlock>>>(dev_a,dev_b,dev_c,width,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i blocks of %i threads : %f \n",blocks,threads,total_time);
	avg_time+=total_time;
	}
	avg_time/=10.0;
	printf("average time for %ix%i matrix sum is %f ",width,width,avg_time);
	//copy back and prints
	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n a[%i]=%i\n",idx,c[idx]);
 
	//free
	free(a);
	free(b);
	free(c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	return 0;
}



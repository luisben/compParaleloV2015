
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecIncrease(int *a, int amount, int length){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < length)
		a[tid]+=amount;
}

int main(int argc, char* argv[]){

	//generic initialization code
	int size,threads,blocks,totalSize;
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	if(argv[2])
		size = atoi(argv[2]);
	else 
		size = 65536;
	if(argv[1])
		threads = atoi(argv[1]);
	else
		threads = 64;
	blocks = (size/threads);
	if(blocks*threads != size)
		blocks++;
	totalSize = size*sizeof(int);
	printf("\n%i blocks of %i threads = %i threads total \n",blocks,threads,blocks*threads);

	//end generic
	//start mallocs
	int *a,*dev_a;

	hipMalloc((void**)&dev_a,totalSize);
	a = (int*) malloc(totalSize);
	//end mallocs

	//problem specific
	int idx,incr_amount;
	incr_amount=5;

	for(idx=0;idx<size;idx++){
		a[idx] = idx;
	}
	
	//copy to dev
	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	//end copy
	
	//call kernel and measure times
	hipEventRecord(start,0);
	vecIncrease<<<blocks,threads>>>(dev_a,incr_amount,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i blocks of %i threads : %f \n",blocks,threads,total_time);
	
	//copy back and prints
	hipMemcpy(a,dev_a,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n a[%i]=%i\n",idx,a[idx]);
 
	//free
	free(a);

	hipFree(dev_a);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}



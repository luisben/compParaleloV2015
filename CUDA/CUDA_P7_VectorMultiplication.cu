
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecMult(int *a, int *b, int *c, int length){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < length)
		c[tid] = a[tid] * b[tid];
}

int main(int argc, char* argv[]){

	//initialization code
	int size,threads,blocks,totalSize;
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	if(argv[2])
		size = atoi(argv[2]);
	else 
		size = 65536;
	if(argv[1])
		threads = atoi(argv[1]);
	else
		threads = 64;
	blocks = (size/threads);
	if(blocks*threads != size)
		blocks++;
	totalSize = size*sizeof(int);
	printf("\n%i blocks of %i threads = %i threads total \n",blocks,threads,blocks*threads);
	//end init
	
	//start mallocs
	int *a,*dev_a,*b,*dev_b,*c,*dev_c;

	hipMalloc((void**)&dev_a,totalSize);
	hipMalloc((void**)&dev_b,totalSize);
	hipMalloc((void**)&dev_c,totalSize);

	a = (int*) malloc(totalSize);
	b = (int*) malloc(totalSize);
	c = (int*) malloc(totalSize);
	//end mallocs

	//problem specific
	int idx;

	for(idx=0;idx<size;idx++){
		a[idx] = idx;
		b[idx] = idx*2;
	}
	
	//copy to dev
	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,totalSize,hipMemcpyHostToDevice);
	//end copy
	
	int iteration = 0;
	float avg_time = 0.0;
	for(iteration=0;iteration<10;iteration++){
	//call kernel and measure times
	hipEventRecord(start,0);
	vecMult<<<blocks,threads>>>(dev_a,dev_b,dev_c,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i blocks of %i threads : %f \n",blocks,threads,total_time);
	avg_time+=total_time;
	}
	avg_time/=10.0;
	printf("average time for %i size vector mult is %f ",size,avg_time);
	//copy back and prints
	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n a[%i]=%i\n",idx,c[idx]);
 
	//free
	free(a);
	free(b);
	free(c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	return 0;
}




#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 4194304
#define THREADS 64

__global__ void vecAdd(int *a, int *b, int *c){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < N )
		c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]){

	int *a,*b,*c;
	int *dev_a,*dev_b,*dev_c;
	int totalSize = N*sizeof(int);
	int idx;
	int size,blocks,threads;
	
	
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	size = N;
	blocks = size/THREADS;
	threads = THREADS;

	hipMalloc((void**)&dev_a,totalSize);
	hipMalloc((void**)&dev_b,totalSize);
	hipMalloc((void**)&dev_c,totalSize);

	a = (int*) malloc(totalSize);
	b = (int*) malloc(totalSize);
	c = (int*) malloc(totalSize);
	
	for(idx=0;idx<N;idx++){
		a[idx] = idx;
		b[idx] = idx*2;
	}

	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,totalSize,hipMemcpyHostToDevice);

	
	int iteration = 0;
	float avg_time = 0.0;
	for(iteration=0;iteration<10;iteration++){
	//call kernel and measure times
	hipEventRecord(start,0);
	vecAdd<<<blocks,threads>>>(dev_a,dev_b,dev_c);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for %i blocks of %i threads : %f \n",blocks,threads,total_time);
	avg_time+=total_time;
	}
	avg_time/=10.0;
	printf("average time for %i size vector mult is %f ",size,avg_time);
	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);
/*
	for(idx=0;idx<N;idx++)
		printf("\n%i+%i=%i\n",a[idx],b[idx],c[idx]);
*/
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}



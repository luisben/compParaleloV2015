
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void vecAdd(int *a, int *b, int *c){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < N )
		c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]){

	int a[N],b[N],c[N];
	int *dev_a,*dev_b,*dev_c;
	int totalSize = N*sizeof(int);
	int idx;

	hipMalloc((void**)&dev_a,totalSize);
	hipMalloc((void**)&dev_b,totalSize);
	hipMalloc((void**)&dev_c,totalSize);

	for(idx=0;idx<N;idx++){
		a[idx] = idx;
		b[idx] = idx*2;
	}

	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,totalSize,hipMemcpyHostToDevice);

	vecAdd<<<2,5>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);

	for(idx=0;idx<N;idx++)
		printf("\n%i+%i=%i\n",a[idx],b[idx],c[idx]);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void vecAdd(int *a, int *b, int *c, int length){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if(tid < length)
		c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]){

	int size = 16384;
	int *a,*b,*c;
	int *dev_a,*dev_b,*dev_c;
	int totalSize = size*sizeof(int);
	int idx;
	//timemeasure
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time_128,time_256,time_512;
	
	hipMalloc((void**)&dev_a,totalSize);
	hipMalloc((void**)&dev_b,totalSize);
	hipMalloc((void**)&dev_c,totalSize);

	a = (int*) malloc(totalSize);
	b = (int*) malloc(totalSize);
	c = (int*) malloc(totalSize);
	
	for(idx=0;idx<size;idx++){
		a[idx] = idx;
		b[idx] = idx+1;
	}

	hipMemcpy(dev_a,a,totalSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,totalSize,hipMemcpyHostToDevice);
	
	hipEventRecord(start,0);
	vecAdd<<<512,32>>>(dev_a,dev_b,dev_c,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_512,start,stop);
	printf("time for 512 blocks of 32 threads : %f",time_512);
	
	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n%i+%i=%i\n",a[idx],b[idx],c[idx]);
	
	hipEventRecord(start,0);
	vecAdd<<<256,64>>>(dev_a,dev_b,dev_c,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_256,start,stop);
	printf("time for 256 blocks of 64 threads : %f",time_256);
	
	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n%i+%i=%i\n",a[idx],b[idx],c[idx]);
	
	hipEventRecord(start,0);
	vecAdd<<<128,128>>>(dev_a,dev_b,dev_c,size);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_128,start,stop);
	printf("time for 128 blocks of 128 threads : %f",time_128);

	hipMemcpy(c,dev_c,totalSize,hipMemcpyDeviceToHost);
	for(idx=0;idx<size;idx+=size/5)
		printf("\n%i+%i=%i\n",a[idx],b[idx],c[idx]);

	free(a);
	free(b);
	free(c);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return 0;
}



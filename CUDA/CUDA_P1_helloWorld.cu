
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holaCUDA(float e) {
	printf("Hola, soy el hilo %i del bloque %i con valor pi -> %f \n",threadIdx.x,blockIdx.x,e);
}

int main(int argc, char **argv){
	holaCUDA<<<3,4>>>(3.1416);
	hipDeviceReset();
	return 0;
}
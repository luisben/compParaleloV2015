
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void makeAdjMat(int *csizes, int *cstarts, int *cases, int *adjMat, int matSize, int dataSize){

        int my_case_idx,my_case_size,my_case_start,idx_inner,mat_pos,idx;
        my_case_idx = blockIdx.x*blockDim.x + threadIdx.x;
        if(my_case_idx < dataSize){
            my_case_size = csizes[my_case_idx];
            my_case_start = cstarts[my_case_idx];
            for(idx=my_case_start;idx<(my_case_start+my_case_size);idx++){
                for(idx_inner=my_case_start;idx_inner<(my_case_start+my_case_size);idx_inner++){
                    mat_pos = (cases[idx] - 1)*matSize + (cases[idx_inner] - 1);
                    adjMat[mat_pos]++;
                }
            }
            } 
}

void genAdjMat(int *case_sizes, int *cases_afecs, int *case_count, int *input_size, int *adjMatrix, int *adjMatDim){

	int num_cases = case_count[0];
	int num_inputs = input_size[0];
	int mat_size = adjMatDim[0];
	
	float total_time;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int case_count_totsize = num_cases * sizeof(int);
	int mat_totsize = mat_size * mat_size * sizeof(int);
	int input_totsize = num_inputs * sizeof(int);

    int *case_stidx = (int*) malloc(case_count_totsize);
	
	int *dev_sizes,*dev_starts,*dev_cases,*dev_matrix;
	
	hipMalloc((void**)&dev_sizes,case_count_totsize);
	hipMalloc((void**)&dev_starts,case_count_totsize);
	hipMalloc((void**)&dev_cases,input_totsize);
	hipMalloc((void**)&dev_matrix,mat_totsize);
	
	hipMemset(dev_matrix,0,mat_totsize);
	
    int idx = 0;
    
	case_stidx[0] = 0;
	for(idx=1;idx<num_cases;idx++){
		case_stidx[idx] = case_stidx[idx-1]+case_sizes[idx-1];
	}

	hipMemcpy(dev_sizes,case_sizes,case_count_totsize,hipMemcpyHostToDevice);
	hipMemcpy(dev_starts,case_stidx,case_count_totsize,hipMemcpyHostToDevice);
	hipMemcpy(dev_cases,cases_afecs,input_totsize,hipMemcpyHostToDevice);

        int blockSize = 32;
        int blockCount = num_inputs/blockSize;
        while(blockCount*blockSize < num_inputs)
            blockCount++;

	hipEventRecord(start,0);
	makeAdjMat<<<blockCount,blockSize>>>(dev_sizes,dev_starts,dev_cases,dev_matrix,mat_size,num_cases);
		
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&total_time,start,stop);
	printf("\n time for kernel processing %i threads in %i blocks : %f \n",blockCount*blockSize,blockCount,total_time);	
	
	hipMemcpy(adjMatrix,dev_matrix,mat_totsize,hipMemcpyDeviceToHost);

    free(case_stidx);
	
	hipFree(dev_sizes);
	hipFree(dev_starts);
	hipFree(dev_cases);
	hipFree(dev_matrix);

}


int main(){

	int cases_sizes[1459] = {2,3,3,3,2,3,2,2,2,2,2,2,2,4,2,2,2,2,4,2,2,2,2,2,2,2,2,3,2,2,2,3,2,5,5,2,2,2,2,2,3,2,2,3,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,4,2,3,3,2,2,3,5,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,3,2,5,2,2,3,3,2,3,2,2,2,2,3,3,2,2,2,2,2,3,2,3,2,3,3,2,2,2,2,2,2,3,3,2,2,3,2,2,2,2,2,2,3,2,2,2,2,2,2,2,3,2,2,2,2,2,3,2,3,2,2,2,2,2,2,4,3,3,2,3,3,2,2,2,2,3,2,3,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,3,4,4,3,3,2,2,2,2,2,2,2,2,2,2,2,3,2,2,2,2,3,3,2,3,2,3,2,2,2,2,2,2,3,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,4,2,3,2,2,2,2,2,2,2,2,2,2,3,3,2,2,2,3,2,2,2,3,3,2,2,3,2,2,3,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,3,2,2,3,3,3,2,2,2,3,2,2,2,2,2,2,2,2,3,2,2,3,2,2,2,2,3,2,2,2,2,4,5,2,2,2,4,2,2,2,2,2,2,2,2,2,2,3,2,2,2,2,3,2,2,2,2,2,2,2,3,3,3,3,2,2,3,2,2,2,2,2,3,2,2,2,3,2,2,2,2,2,2,2,6,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,2,2,2,3,3,3,2,2,2,2,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,3,2,3,2,2,2,2,2,2,2,2,2,4,3,2,4,2,2,2,2,2,2,2,4,2,2,3,2,3,4,2,2,2,3,2,2,3,2,2,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,3,2,2,2,2,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,3,3,2,3,3,2,2,2,2,2,3,2,2,2,3,2,2,3,2,2,2,2,3,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,3,2,3,2,2,2,2,2,3,2,2,2,2,2,2,2,2,2,2,2,2,2,3,2,3,2,2,2,2,2,2,2,2,2,2,3,2,2,2,2,2,2,2,3,2,2,3,2,3,2,2,1};
	int cases_data[2222] = {140,244,55,183,62,271,81,80,272,171,55,146,167,272,173,184,12,84,55,27,185,159,153,147,4,271,183,271,27,58,201,272,27,123,271,108,138,272,36,167,185,55,55,83,29,271,276,272,73,276,271,276,157,271,271,276,271,276,140,276,145,276,271,276,276,276,271,271,271,276,276,276,27,60,271,271,35,50,34,271,276,272,276,156,34,159,271,7,23,55,250,272,2,37,171,200,20,55,272,167,137,55,8,272,153,40,167,153,237,158,272,183,173,115,36,98,91,271,117,27,87,34,272,272,159,272,272,55,166,20,234,27,183,272,184,159,65,55,9,198,198,13,62,53,138,272,218,159,272,275,32,77,95,51,171,272,272,183,171,249,118,246,184,140,167,195,62,203,159,157,46,27,97,30,114,167,198,55,272,167,183,27,238,20,36,20,119,277,167,272,147,271,233,27,24,271,192,119,272,55,63,22,21,146,96,246,38,147,271,180,276,181,272,276,198,157,271,197,198,272,272,57,272,198,86,276,276,180,180,276,271,276,276,271,276,276,271,271,271,271,271,276,271,271,181,271,271,171,180,276,276,276,171,271,276,158,276,276,119,159,167,161,198,198,171,37,272,18,198,194,20,123,272,272,55,272,198,271,119,123,198,195,181,151,55,55,125,116,177,182,198,271,272,272,198,198,134,153,272,271,271,272,276,271,276,271,276,276,276,276,276,272,271,156,276,180,271,158,271,276,276,276,271,276,171,272,271,198,272,195,272,274,194,171,272,195,271,195,272,194,251,167,167,276,271,271,276,180,276,271,276,175,271,276,271,276,271,271,140,181,276,272,276,272,276,171,171,276,180,276,271,271,276,180,271,180,272,271,276,276,276,276,271,276,271,276,171,276,171,180,276,271,276,167,271,271,271,276,276,163,272,276,271,276,271,276,276,276,271,276,271,276,276,276,271,276,180,145,138,276,271,180,276,271,271,271,276,171,171,271,271,150,276,276,271,276,276,276,271,276,157,272,167,276,276,272,271,150,276,272,195,183,198,272,158,198,167,272,276,138,180,276,272,272,271,276,271,271,272,195,180,271,271,276,198,183,276,276,145,271,276,156,180,271,276,198,271,198,276,271,271,156,276,276,276,276,276,276,276,276,276,271,276,271,276,276,276,272,271,271,271,271,272,271,180,180,271,272,271,276,271,276,276,276,7,271,272,272,138,141,272,141,201,272,201,201,201,171,272,201,272,276,276,271,276,272,276,145,276,272,157,276,276,276,272,271,276,271,271,203,271,180,276,276,272,150,271,271,271,276,145,276,271,271,276,276,276,271,276,271,180,276,271,180,271,271,271,180,276,276,180,271,180,276,271,276,271,276,272,27,271,271,276,276,180,276,272,276,88,276,20,183,271,62,171,56,191,159,140,140,191,201,171,138,150,171,171,272,146,272,171,272,27,171,171,20,167,171,16,158,271,271,271,211,190,55,222,73,136,27,26,177,55,271,276,271,276,276,276,180,276,180,271,271,271,180,180,271,276,276,271,276,276,271,271,271,276,271,276,271,276,276,276,272,276,180,276,276,276,276,180,271,272,271,276,27,271,271,171,271,276,180,272,271,171,201,272,204,198,272,276,271,39,216,123,158,272,20,236,201,78,201,272,185,171,159,167,201,104,17,125,89,159,159,141,50,218,223,272,272,167,213,272,272,141,272,271,185,272,27,55,34,55,146,213,201,20,171,271,159,272,201,271,159,272,271,272,23,55,201,272,272,20,201,201,44,272,272,128,141,185,159,171,217,235,106,185,224,55,245,171,157,180,276,272,157,276,276,271,276,272,145,271,271,276,272,276,272,276,276,272,180,271,276,271,271,276,276,276,272,272,11,276,276,271,180,276,276,276,271,276,272,271,272,113,151,85,271,158,271,271,271,43,34,171,191,272,271,276,276,27,271,271,180,276,209,141,183,271,167,272,112,201,153,77,178,201,171,140,101,185,271,272,140,144,138,272,171,171,271,246,93,72,69,276,3,271,276,271,180,276,276,272,271,272,276,276,276,139,271,271,276,276,276,271,276,276,55,276,271,271,271,271,272,276,272,276,271,271,271,251,201,27,276,276,180,276,276,271,276,271,271,271,271,271,271,276,276,271,276,271,103,276,271,276,276,272,276,181,271,276,33,271,276,271,272,276,181,128,276,276,158,276,23,146,73,64,27,201,119,55,62,62,271,271,276,276,272,271,271,23,276,276,272,276,55,230,276,33,55,62,119,31,5,271,219,276,4,76,28,111,20,48,256,276,272,276,271,271,276,271,271,276,276,276,271,108,42,194,276,269,271,271,183,276,42,27,41,272,123,140,15,271,276,70,27,119,276,271,138,20,30,66,168,271,195,119,135,272,276,276,55,114,36,140,119,31,272,77,181,27,272,108,119,29,74,276,271,55,271,20,271,276,272,276,276,272,271,272,271,271,276,276,195,123,276,272,271,154,121,271,195,167,159,272,272,198,159,276,276,271,272,276,271,276,271,276,123,276,276,276,276,276,276,276,14,276,276,14,276,276,276,145,133,276,146,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,272,131,134,276,276,276,276,276,276,276,276,271,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,272,276,276,276,276,276,276,272,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,276,79,198,271,272,146,107,100,276,89,146,276,61,198,30,96,260,271,20,271,55,195,272,158,255,110,272,146,272,257,158,271,55,272,158,158,271,93,271,271,271,153,185,158,158,159,167,271,272,191,158,146,147,271,157,183,272,271,159,271,272,272,159,159,183,155,271,171,271,171,155,271,167,167,271,159,271,182,192,171,159,146,142,171,185,158,271,271,188,271,159,271,171,167,271,184,168,271,271,242,158,183,271,184,158,271,138,171,176,271,271,271,171,171,271,272,207,138,271,171,271,153,194,271,150,271,171,158,202,271,178,138,177,198,173,271,272,167,198,272,271,272,146,247,271,272,272,185,271,272,143,271,271,185,194,198,167,271,271,171,271,271,198,171,132,271,271,167,271,159,171,271,194,198,198,138,272,198,272,198,158,183,140,272,190,191,271,272,140,251,159,272,191,271,171,140,184,153,137,147,271,272,171,166,271,184,142,153,171,271,271,271,185,198,195,198,271,271,167,178,272,271,171,171,140,171,272,272,271,271,138,271,138,152,271,171,271,184,271,198,198,271,271,192,272,271,206,138,272,171,272,130,272,271,272,146,262,272,171,171,158,271,182,204,171,192,271,171,272,271,241,185,158,146,140,167,252,171,198,198,198,271,271,271,30,123,214,2,271,271,271,198,171,271,146,272,272,167,164,271,171,201,185,67,271,198,138,153,171,25,271,147,167,167,272,272,198,198,271,171,171,159,157,159,158,146,271,158,271,79,198,271,271,195,185,271,25,273,90,27,192,271,272,220,246,271,195,146,171,171,271,140,171,1,279,198,271,271,272,272,158,71,27,198,271,198,271,140,171,271,171,271,158,79,271,272,182,271,272,171,29,271,279,272,55,59,129,271,122,135,272,199,248,225,271,78,261,272,160,171,279,171,207,25,272,228,44,201,201,272,138,272,25,272,271,141,201,201,272,272,279,271,264,272,201,272,272,272,271,272,271,201,147,272,44,272,272,272,271,219,171,201,55,271,55,272,201,27,272,271,271,272,141,201,271,171,272,27,272,272,167,158,272,27,201,272,201,271,272,271,55,272,272,271,27,272,78,271,208,272,271,272,272,272,201,272,272,187,248,271,141,271,271,272,272,272,272,272,212,279,272,201,101,272,272,251,271,221,272,272,141,272,171,271,272,169,10,272,167,254,272,27,27,272,102,201,272,215,271,189,272,218,272,27,272,74,272,272,124,272,272,272,272,216,186,271,272,141,271,272,272,27,271,231,141,272,171,271,249,265,272,268,272,279,164,272,201,272,271,243,141,201,272,272,272,272,94,272,201,271,167,201,266,219,272,73,201,272,201,272,141,201,167,272,272,272,272,232,240,218,272,272,272,272,272,201,272,272,272,249,105,272,271,272,127,201,201,201,272,47,147,239,201,75,54,272,272,271,201,271,27,272,210,20,201,272,230,52,87,272,272,201,271,271,271,272,250,128,271,82,6,217,83,272,272,272,227,128,226,218,271,146,241,55,49,193,272,272,73,20,250,40,272,109,272,272,271,272,272,55,271,27,272,271,218,73,272,146,89,39,55,253,147,243,73,96,272,272,271,141,38,272,221,201,148,271,271,265,57,2,36,55,55,272,45,224,272,277,119,19,271,272,271,271,271,272,272,272,272,92,272,271,272,114,272,272,272,272,272,229,272,278,272,272,272,118,272,272,276,271,271,271,276,276,276,276,271,276,271,276,271,271,271,272,276,276,276,140,271,200,272,198,194,276,276,198,194,159,276,146,194,158,159,162,196,270,58,159,157,259,185,258,267,272,159,272,138,140,20,276,276,276,276,271,271,271,271,271,157,167,180,272,276,271,276,271,276,271,276,145,271,276,271,276,180,276,272,276,271,271,276,150,180,276,276,271,271,276,271,180,271,276,272,155,271,181,208,271,171,167,120,276,272,7,200,138,276,276,55,276,272,271,271,180,99,185,158,272,276,271,276,207,179,140,68,272,272,103,276,276,171,138,271,55,276,176,185,163,271,138,159,276,159,167,140,171,276,165,174,178,123,138,272,272,27,272,170,171,272,126,137,158,171,132,172,272,185,159,167,205,140,55,271,159,271,272,156,271,103,276,272,271,2,276,119,138,171,271,171,271,3,27,88,89,123,263,119,167,159,272,20,123,272,158,264,61,183,146,34,138,271,171,191,171,271,171,271,181,158,157,272,204,185,191,155,271,153,182,271,272,171,20,263,178,138,140,264,198,272,159,185,27,150,44,271,158,272,167,271,159,171,157,149,55,272,171,158,171,159,272,171,272,171,146,272};
	int case_count[1] = {1459};
    int input_size[1] = {2222};
    int adj_mat_size[1] = {279};
    int adj_mat[279*279] = {0};
    genAdjMat(cases_sizes,cases_data,case_count,input_size,adj_mat,adj_mat_size);
    int idx,row_count=0;
	FILE *f = fopen("adjmat.txt","w");
	if(f==0)
		exit(1);
	
    for(idx=0;idx<adj_mat_size[0]*adj_mat_size[0];idx++){
        if(idx%adj_mat_size[0]==0){
            fprintf(f,"\n%i",row_count);
            row_count++;
        }
        fprintf(f,"| %i |",adj_mat[idx]);
    }
	fclose(f);
}
